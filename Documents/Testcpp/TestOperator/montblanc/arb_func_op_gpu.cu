
#include <hip/hip_runtime.h>
#if GOOGLE_CUDA

#include "arb_func_op_gpu.cuh"

ARBFUNC_NAMESPACE_BEGIN
ARBFUNC_ARB_FUNC_NAMESPACE_BEGIN


// Register a GPU kernel for ArbFunc
// handling permutation ['float']
REGISTER_KERNEL_BUILDER(
    Name("ArbFunc")
    .TypeConstraint<float>("FT")
    .Device(tensorflow::DEVICE_GPU),
    ArbFunc<GPUDevice, float>);

// Register a GPU kernel for ArbFunc
// handling permutation ['double']
REGISTER_KERNEL_BUILDER(
    Name("ArbFunc")
    .TypeConstraint<double>("FT")
    .Device(tensorflow::DEVICE_GPU),
    ArbFunc<GPUDevice, double>);



ARBFUNC_ARB_FUNC_NAMESPACE_STOP
ARBFUNC_NAMESPACE_STOP

#endif // #if GOOGLE_CUDA